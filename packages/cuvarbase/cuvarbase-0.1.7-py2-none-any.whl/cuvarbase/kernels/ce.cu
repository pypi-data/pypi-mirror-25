#include "hip/hip_runtime.h"
#include <stdio.h>

//{CPP_DEFS}

#ifdef DOUBLE_PRECISION
	#define ATOMIC_ADD atomicAddDouble
	#define FLT double
#else
	#define ATOMIC_ADD atomicAdd
	#define FLT float
#endif

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__device__ int phase_ind(FLT ft){
	FLT phi = ft - floor(ft);
	int n = (int) (phi * NPHASE);
	return n % NPHASE;
}

__device__ int posmod(int n, int N){
	int nmodN = n % N;
	return (nmodN < 0) ? nmodN + N : nmodN;
}

__global__ void histogram_data_weighted(FLT *t, FLT *y, FLT *dy, FLT *bin, FLT *freqs,
	                               int nfreq, int ndata, FLT max_phi){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int i_freq = i / ndata;
	int j_data = i % ndata;

	if (i_freq < nfreq && j_data < ndata){
		FLT Y = y[j_data];
		FLT DY = dy[j_data];
		
		int n0 = phase_ind(freqs[i_freq] * t[j_data]);
		int offset = i_freq * (NMAG * NPHASE);

		int m0 = (int) (Y * NMAG);

		for(int m = 0; m < NMAG; m++){
			FLT z = (((FLT) m) / NMAG - Y);
			if (abs(z) > max_phi * DY && m != m0)
				continue;
			FLT zmax = z + (1 + MAG_OVERLAP) / ((FLT) NMAG);
			FLT wtot = normcdf(zmax / DY) - normcdf(z / DY);

			//if (wtot > 1E-2)
			//	printf("%e %e %e %e %e\n", wtot, z, zmax, Y, DY);
			for(int n = n0; n >= n0 - PHASE_OVERLAP; n--)
				ATOMIC_ADD(&(bin[offset + posmod(n, NPHASE) * NMAG + m]), wtot);
			
		}
	}

}

__global__ void histogram_data_count(FLT *t, unsigned int *y, unsigned int *bin, FLT *freqs,
	                                 int nfreq, int ndata){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int i_freq = i / ndata;
	int j_data = i % ndata;

	if (i_freq < nfreq && j_data < ndata){
		int offset = i_freq * (NMAG * NPHASE);
		unsigned int m0 = y[j_data];
		int n0 = phase_ind(freqs[i_freq] * t[j_data]);

		for (int n = n0; n >= n0 - PHASE_OVERLAP; n--){
			for (int m = m0; m >= 0 && m >= m0 - MAG_OVERLAP; m--) {
				atomicInc(&(bin[offset + posmod(n, NPHASE) * NMAG + m]), 
				      (PHASE_OVERLAP + 1) * (MAG_OVERLAP + 1) * ndata);
			}
		}	
	}
}


__global__ void weighted_ce(FLT *bins, int nfreq, FLT *ce){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT Hc = 0.f;
		FLT bin_tot = 0.f;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			FLT p_phi_n = 0.f;
			for (int m=0; m < NMAG; m++)
				p_phi_n += bins[offset + m];

			for (int m=0; m < NMAG; m++){
				FLT pmn = bins[offset + m];
				bin_tot += pmn;

				if (pmn > 0.f && p_phi_n > 1E-10)
					Hc += pmn * log(p_phi_n / pmn);
			}
		}
		ce[i] = Hc / bin_tot;
	}
}

__global__ void standard_ce(unsigned int *bins, int nfreq,
                                    FLT *ce){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT Hc = 0.f;
		unsigned int bin_tot = 0;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			unsigned int p_phi_n = 0;
			for (int m=0; m < NMAG; m++)
				p_phi_n += bins[offset + m];

			for (int m=0; m < NMAG; m++){
				FLT pmn = bins[offset + m];
				bin_tot += pmn;

				if (pmn > 0 && p_phi_n > 0)
					Hc += pmn * log(((FLT) p_phi_n) / ((FLT) pmn));
			}
		}
		
		ce[i] = Hc / bin_tot;
	}
}