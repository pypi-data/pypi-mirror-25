#include "hip/hip_runtime.h"
#include <stdio.h>

//{CPP_DEFS}

#ifndef MAX_SHARED_MEM_SIZE
	#define MAX_SHARED_MEM_SIZE 48000
#endif

#ifdef DOUBLE_PRECISION
	#define ATOMIC_ADD atomicAddDouble
	#define FLT double
#else
	#define ATOMIC_ADD atomicAdd
	#define FLT float
#endif

#define MAX_SHARED_FLT_SIZE (int) floor(((FLT) MAX_SHARED_MEM_SIZE) / sizeof(FLT))
#define LOCAL_HIST_SIZE \
      NMAG * NPHASE > MAX_SHARED_FLT_SIZE ? MAX_SHARED_FLT_SIZE - 1 : NMAG * NPHASE


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                       (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__device__ int phase_ind(FLT ft){
	FLT phi = ft - floor(ft);
	int n = (int) (phi * NPHASE);
	return n % NPHASE;
}

__device__ int posmod(int n, int N){
	int nmodN = n % N;
	return (nmodN < 0) ? nmodN + N : nmodN;
}

__global__ void histogram_data_weighted(FLT *t, FLT *y, FLT *dy, 
	                                    FLT *bin, FLT *freqs,
	                                    int nfreq, int ndata, 
	                                    FLT max_phi){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int i_freq = i / ndata;
	int j_data = i % ndata;

	if (i_freq < nfreq && j_data < ndata){
		FLT Y = y[j_data];
		FLT DY = dy[j_data];
		
		int n0 = phase_ind(freqs[i_freq] * t[j_data]);
		int offset = i_freq * (NMAG * NPHASE);

		int m0 = (int) (Y * NMAG);

		for(int m = 0; m < NMAG; m++){
			FLT z = (((FLT) m) / NMAG - Y);
			if (abs(z) > max_phi * DY && m != m0)
				continue;
			FLT zmax = z + (1 + MAG_OVERLAP) / ((FLT) NMAG);
			FLT wtot = normcdf(zmax / DY) - normcdf(z / DY);

			//if (wtot > 1E-2)
			//	printf("%e %e %e %e %e\n", wtot, z, zmax, Y, DY);
			for(int n = n0; n >= n0 - PHASE_OVERLAP; n--)
				ATOMIC_ADD(&(bin[offset + posmod(n, NPHASE) * NMAG + m]), wtot);
			
		}
	}

}

__global__ void histogram_data_count(FLT *t, unsigned int *y,
	                                 unsigned int *bin,
	                                 FLT *freqs, int nfreq, 
	                                 int ndata){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int i_freq = i / ndata;
	int j_data = i % ndata;

	if (i_freq < nfreq && j_data < ndata){
		int offset = i_freq * (NMAG * NPHASE);
		unsigned int m0 = y[j_data];
		int n0 = phase_ind(freqs[i_freq] * t[j_data]);

		for (int n = n0; n >= n0 - PHASE_OVERLAP; n--){
			for (int m = m0; m >= 0 && m >= m0 - MAG_OVERLAP; m--) {
				atomicInc(&(bin[offset + posmod(n, NPHASE) * NMAG + m]), 
				      (PHASE_OVERLAP + 1) * (MAG_OVERLAP + 1) * ndata);
			}
		}	
	}
}


__global__ void weighted_ce(FLT *bins, int nfreq, FLT *ce){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT Hc = 0.f;
		FLT bin_tot = 0.f;
		FLT dm = ((FLT)(MAG_OVERLAP + 1)) / NMAG;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			FLT p_phi_n = 0.f;
			for (int m=0; m < NMAG; m++)
				p_phi_n += bins[offset + m];

			for (int m=0; m < NMAG; m++){
				FLT pmn = bins[offset + m];
				bin_tot += pmn;

				if (pmn > 0.f && p_phi_n > 1E-10)
					Hc += pmn * log((dm * p_phi_n) / pmn);
			}
		}
		ce[i] = Hc / bin_tot;
	}
}

__global__ void standard_ce(unsigned int *bins, int nfreq,
                            FLT *ce){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT Hc = 0.f;
		FLT dm = ((FLT)(MAG_OVERLAP + 1)) / NMAG;
		unsigned int bin_tot = 0;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			unsigned int Nphi = 0;
			for (int m=0; m < NMAG; m++)
				Nphi += bins[offset + m];

			if (Nphi == 0)
				continue;

			for (int m=0; m < NMAG; m++){
				unsigned int N = bins[offset + m];

				if (N == 0)
					continue;

				bin_tot += N;
				Hc += N * log((dm * Nphi) / N);
			}
		}
		
		ce[i] = Hc / bin_tot;
	}
}

__global__ void constdpdm_ce(unsigned int *bins, int nfreq,
                             FLT *ce, FLT *mag_bwf){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT Hc = 0.f;
		unsigned int bin_tot = 0;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			unsigned int Nphi = 0;
			for (int m=0; m < NMAG; m++)
				Nphi += bins[offset + m];
			
			if (Nphi == 0)
				continue;

			for (int m=0; m < NMAG; m++){
				unsigned int N = bins[offset + m];

				if (N == 0)
					continue;
				
				bin_tot += N;
				Hc += N * log((mag_bwf[m] * Nphi) / N);
			}
		}
		
		ce[i] = Hc / bin_tot;
	}
}

__global__ void log_prob(unsigned int *bins, int nfreq,
                         FLT *log_proba, FLT *mag_bin_fracs){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nfreq){
		FLT logP = 0.f;
		for(int n=0; n < NPHASE; n++){
			int offset = i * (NMAG * NPHASE) + n * NMAG;

			unsigned int Nphi = 0;
			for (int m=0; m < NMAG; m++)
				Nphi += bins[offset + m];
			
			if (Nphi == 0)
				continue;

			for (int m=0; m < NMAG; m++){
				FLT N = (FLT) (bins[offset + m]);

				FLT Nexp = Nphi * mag_bin_fracs[m];

				if (Nexp < 1e-9)
					continue;

				logP += N * log(Nexp) - Nexp - lgamma(N + 1.f);
			}
		}
		
		log_proba[i] = logP / (PHASE_OVERLAP + 1.f);
	}
}

