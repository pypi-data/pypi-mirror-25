#include "hip/hip_runtime.h"
#include <stdio.h>
#define RESTRICT __restrict__
#define CONSTANT const
#define MIN_W 1E-3
//{CPP_DEFS}

__device__ int get_id(){
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int mod(int a, int b){
	int r = a % b;
	return (r < 0) ? r + b : r;
}

__device__ float mod1(float a){
	return a - floorf(a);
}


__global__ void binned_bls_bst(float *yw, float *w, float *bls, int n){
	int i = get_id();

	if (i < n){
		float wtot = w[i];
		float ybar = yw[i];

		bls[i] = (wtot > 1e-10 && wtot < 1.f - 1e-10) ?
					ybar * ybar / (wtot * (1.f - wtot)) : 0.f;
		
	}
}

__global__ void store_best_sols(int *argmaxes, float *best_phi, 
	                            float *best_q,
	                            int nbins0, int nbinsf, int noverlap, 
	                            float dlogq, int nfreq, int freq_offset){

	int i = get_id();

	if (i < nfreq){
		int imax = argmaxes[i + freq_offset];
		float dphi = 1.f / noverlap;
		int nb = nbins0;
		float x = 1.f;
		int offset = 0;

		while(offset + noverlap * nb <= imax){	
			offset += noverlap * nb;

			x *= (1 + dlogq);
			nb = (int) (x * nbins0);
		}

		float q = 1.f / nb;
		int s = (imax - offset) / nb;

		int jphi = (imax - offset) % nb;
		
		float phi = mod1(q * (jphi + s * dphi));

		best_phi[i + freq_offset] = phi;
		best_q[i + freq_offset] = q;
	}
}

__global__ void store_best_sols_custom(int *argmaxes, float *best_phi, 
	                            float *best_q, float *q_values,
	                            float *phi_values, int nq, int nphi,
	                            int nfreq, int freq_offset){

	int i = get_id();

	if (i < nfreq){
		int imax = argmaxes[i + freq_offset];

		best_phi[i + freq_offset] = phi_values[imax / nq];
		best_q[i + freq_offset] = q_values[imax % nq];
	}
}


// needs ndata * nfreq threads
// noverlap -- number of overlapped bins (noverlap * (1 / q) total bins)
// Note: this thread heavily utilizes global atomic operations, and could
//       likely be improved by 1-2 orders of magnitude for large Ndata (10^4)
//       if shared memory atomics were utilized.
__global__ void bin_and_phase_fold_bst_multifreq(
	                    float *t, float *yw, float *w,
						float *yw_bin, float *w_bin, float *freqs,
						int ndata, int nfreq, int nbins0, int nbinsf,
						int freq_offset, int noverlap, float dlogq,
						int nbins_tot){
	int i = get_id();

	if (i < ndata * nfreq){
		int i_data = i % ndata;
		int i_freq = i / ndata;

		int offset = i_freq * nbins_tot * noverlap;

		float W = w[i_data];
		float YW = yw[i_data];

		// get phase [0, 1)
		float phi = mod1(t[i_data] * freqs[i_freq + freq_offset]);

		float dphi = 1.f / noverlap;
		int nbtot = 0;

		// iterate through bins (logarithmically spaced)
		for(float x = 1.f; ((int) (x * nbins0)) <= nbinsf; 
			                              x *= (1 + dlogq)){
			int nb = (int) (x * nbins0);
			float q = 1.f / nb;

			// iterate through offsets [ 0, 1./sigma, ..., 
			//                           (sigma - 1) / sigma ]
			for (int s = 0; s < noverlap; s++){

				int b = (int) floorf(nb * mod1(phi - s * q * dphi));

				b += offset + s * nb + noverlap * nbtot;
				atomicAdd(&(yw_bin[b]), YW);
				atomicAdd(&(w_bin[b]), W);
			}
			nbtot += nb;
		}
	}
}


// needs ndata * nfreq threads
// noverlap -- number of overlapped bins (noverlap * (1 / q) total bins)
__global__ void bin_and_phase_fold_custom(
	                    float *t, float *yw, float *w,
						float *yw_bin, float *w_bin, float *freqs,
						float *q_values, float *phi_values, 
						int nq, int nphi, int ndata, 
						int nfreq, int freq_offset){
	int i = get_id();

	if (i < ndata * nfreq){
		int i_data = i % ndata;
		int i_freq = i / ndata;

		int offset = i_freq * nq * nphi;

		float W = w[i_data];
		float YW = yw[i_data];

		// get phase [0, 1)
		float phi = mod1(t[i_data] * freqs[i_freq + freq_offset]);

		for(int pb = 0; pb < nphi; pb++){
			float dphi = phi - phi_values[pb];
			dphi -= floorf(dphi);

			for(int qb = 0; qb < nq; qb++){
				if (dphi < q_values[qb]){
					atomicAdd(&(yw_bin[pb * nq + qb + offset]), YW);
					atomicAdd(&(w_bin[pb * nq + qb + offset]), W);
				}
			}
		}
	}
}


__global__ void reduction_max(float *arr, int *arr_args, int nfreq, 
	                          int nbins, int stride,
                              float *block_max, int *block_arg_max, 
                              int offset, int init){
	__shared__ float partial_max[BLOCK_SIZE];
	__shared__ int partial_arg_max[BLOCK_SIZE];

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	int nblocks_per_freq = gridDim.x / nfreq;
	int nthreads_per_freq = blockDim.x * nblocks_per_freq;


	//	freq_no / b
	//			----block 1 -----       ----- block N ------------------------
	//		  0 | 0 1 2 .. B - 1 | ... | (N - 1)B, ... , ndata, ..., N * B - 1|
	//
	//			---block N + 1---       ---- block 2N ------------------------
	//		  1 | 0 1 2 .. B - 1 | ... | (N - 1)B, ... , ndata, ..., N * B - 1|
	//			...
	//
	//			---(nf - 1)N ----       --- nf * N ---
	//   nf - 1 | ..             | ... |             |

	int fno = id / nthreads_per_freq;
	int b   = id % nthreads_per_freq;

	// read part of array from global memory into shared memory
	partial_max[threadIdx.x] = (fno < nfreq && b < nbins) ?
	                                 arr[fno * stride + b] : -1.f;

	partial_arg_max[threadIdx.x] = (fno < nfreq && b < nbins) ?
									(
										(init == 1) ?
											b : arr_args[fno * stride + b]
									) : -1;

	__syncthreads();

	float m1, m2;

	// reduce to find max of shared memory array
	for(int s = blockDim.x / 2; s > 0; s /= 2){
		if(threadIdx.x < s){
			m1 = partial_max[threadIdx.x];
			m2 = partial_max[threadIdx.x + s];

			partial_max[threadIdx.x] = (m1 > m2) ? m1 : m2;

			partial_arg_max[threadIdx.x] = (m1 > m2) ?
			 						partial_arg_max[threadIdx.x] :
			 						partial_arg_max[threadIdx.x + s];
		}

		__syncthreads();
	}

	// store partial max back into global memory
	if (threadIdx.x == 0 && fno < nfreq){
		int i = (gridDim.x == nfreq) ? 0 :
			fno * stride - fno * nblocks_per_freq;

		i += blockIdx.x + offset;

		block_max[i] = partial_max[0];
		block_arg_max[i] = partial_arg_max[0];
	}
}
